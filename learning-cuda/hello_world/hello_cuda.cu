
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu(){
    printf("Hello Word from the GPU \n");
}


int main(void){
    hello_from_gpu <<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}