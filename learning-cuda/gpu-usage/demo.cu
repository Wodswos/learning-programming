// #include <stdio.h>
// #include <unistd.h>
// #include <stdlib.h>

// const long long tdelay=1000000LL;
// const int loops = 10000;
// const int hdelay = 1;

// __global__ void dkern(){

//     long long start = clock64();
//     while(clock64() < start+tdelay);
// }

// int main(int argc, char *argv[]){

//     int my_delay = hdelay;
//     if (argc > 1) my_delay = atoi(argv[1]);
//     for (int i = 0; i<loops; i++){
//         dkern<<<1,1>>>();
//         usleep(my_delay);}

//     return 0;
// }


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>

const long long tdelay = 1000000LL;
const int loops = 10000;
const int hdelay = 1;

__global__ void dkern() {
    long long start = clock64();
    while (clock64() < start + tdelay);
}

int main(int argc, char *argv[]) {
    int my_delay = hdelay;
    if (argc > 1) {
        char *endptr;
        long val = strtol(argv[1], &endptr, 10);
        if (*endptr != '\0' || val < 0) {
            fprintf(stderr, "Invalid delay value\n");
            return 1;
        }
        my_delay = (int)val;
    }

    for (int i = 0; i < loops; i++) {
        dkern<<<1, 1>>>();
        hipDeviceSynchronize(); // Ensure the kernel execution is complete
        usleep(my_delay);
    }

    return 0;
}