/******************************************************
 * 文件名: multi_device_per_thread.cu
 * refer to https://docs.nvidia.com/deeplearning/nccl/user-guide/docs/examples.html#example-3-multiple-devices-per-thread
 * 编译:  mpicxx -o mdpt multi_device_per_thread.cu -lnccl -lcudart -lcuda
 * 运行:  mpirun -np 2 ./mdpt  (可根据需要修改 -np)
 ******************************************************/


#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include