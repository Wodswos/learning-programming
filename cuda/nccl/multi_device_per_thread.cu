/******************************************************
 * filename: multi_device_per_thread.cu
 * From https://docs.nvidia.com/deeplearning/nccl/user-guide/docs/examples.html#example-3-multiple-devices-per-thread
 * compile & run:  make stmp &&  mpirun -np 2 ./stmd 
 ******************************************************/


#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
