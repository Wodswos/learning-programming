/******************************************************
 * filename: test_nccl.cu
 * compile & run: make test_nccl && mpirun -np 2 ./test_nccl
 ******************************************************/

/*
TODO: 
 * Q: Reduce 操作是否可以用 send buffer 作为 recv buffer ?
 * A: Yes.
 * Q: C++ enumerate 可不可以用对应的值？
 * A: No, argument of type "int" is incompatible with parameter of type "hipMemcpyKind"
*/


#include <mpi.h>
#include <nccl.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define CHECK_CUDA(call)                                                     \
    do{                                                                      \
        hipError_t err = call;                                              \
        if(err != hipSuccess){                                              \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__     \
                      << " code=" << err << "\"" << hipGetErrorString(err)  \
                      << "\"" << std::endl;                                   \
            exit(EXIT_FAILURE);                                               \
        }                                                                    \
    }while(0)

#define CHECK_NCCL(call) \
    do{ \
        ncclResult_t err = call; \
        if (err != ncclSuccess){ \
            std::cerr << "NCCL error at " << __FILE__ << ":" << __LINE__ \
                      << "code=" << err << "\"" << ncclGetErrorString(err) \
                      << "\"" << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    }while(0)

int main(int argc, char* argv[]){
    // 1. init mpi
    MPI_Init(&argc, &argv);

    int world_size = 0;
    int world_rank = 0;

    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);
    
    // 2. init ncclComm
    // get and broadcast ncclUniqueId
    ncclUniqueId id;
    if(world_rank == 0) {
        CHECK_NCCL(ncclGetUniqueId(&id));
    }
    MPI_Bcast((void *) &id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);

    int gpu_id = world_rank;
    CHECK_CUDA(hipSetDevice(gpu_id));

    // ncclComm
    ncclComm_t comm;
    CHECK_NCCL(ncclCommInitRank(&comm, world_size, id, world_rank));

    // 3. prepare communication data/buffer
    const int data_size=8;
    std::vector<float> host_send(data_size, float(world_rank+1));
    std::vector<float> host_recv(data_size, 0.0f);

    float *device_send = nullptr;
    float *device_recv = nullptr;

    CHECK_CUDA(hipMalloc(&device_send, data_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&device_recv, data_size * sizeof(float)));

    CHECK_CUDA(hipMemcpy(device_send, host_send.data(), data_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(device_recv, host_recv.data(), data_size * sizeof(float), hipMemcpyHostToDevice));

    // 4. Communication
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    CHECK_NCCL(ncclAllReduce(
        (const void*)device_send,
        (void*) device_recv,
        // (void*) device_send,
        data_size,
        ncclFloat,
        ncclSum,
        comm,
        stream
    ));
    
    CHECK_CUDA(hipStreamSynchronize(stream));

    // 5. Copy back to host and print
    CHECK_CUDA(hipMemcpy(host_recv.data(), device_recv, data_size * sizeof(float), hipMemcpyDeviceToHost));
    // CHECK_CUDA(hipMemcpy(host_recv.data(), device_send, data_size * sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "[Rank " << world_rank << "] After AllReduce: ";
    for (int i = 0; i < data_size; i++){
        std::cout << host_recv[i] << " ";
    }
    std::cout << std::endl;
    
    // 6. Clear device
    CHECK_CUDA(hipFree(device_send));
    CHECK_CUDA(hipFree(device_recv));
    CHECK_CUDA(hipStreamDestroy(stream));
    ncclCommDestroy(comm);

    MPI_Finalize();
    return 0;
}